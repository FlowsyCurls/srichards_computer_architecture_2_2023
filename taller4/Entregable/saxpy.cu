
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 10000000
#define BLOCK_SIZE 256

__global__ void saxpy_kernel(float a, float *x, float *y, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = a * x[i] + y[i];
    }
}


void saxpy_serial(float a, float* x, float* y, int n)
{
    for (int i = 0; i < n; i++)
    {
        y[i] = a * x[i] + y[i];
    }
}

int main()
{
    float *x, *y, *d_x, *d_y;
    float a = 2.0;
    int size = N * sizeof(float);

    // Allocate memory on host
    x = (float*) malloc(size);
    y = (float*) malloc(size);

    // Initialize vectors on host
    for (int i = 0; i < N; i++)
    {
        x[i] = i;
        y[i] = i * 2;
    }

    // Allocate memory on device
    hipMalloc((void**) &d_x, size);
    hipMalloc((void**) &d_y, size);

    // Copy vectors from host to device
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    // Call serial saxpy
    clock_t t_serial_start = clock();
    saxpy_serial(a, x, y, N);

    clock_t t_serial_end = clock();
    double t_serial = ((double)(t_serial_end - t_serial_start)) / CLOCKS_PER_SEC;

    // Call parallel saxpy
    clock_t t_parallel_start = clock();
    saxpy_kernel<<<(N + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(a, d_x, d_y, N);
    hipDeviceSynchronize();
    clock_t t_parallel_end = clock();
    double t_parallel = ((double)(t_parallel_end - t_parallel_start)) / CLOCKS_PER_SEC;

    // Copy result from device to host
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

    // Free memory on device
    hipFree(d_x);
    hipFree(d_y);

    // Print execution times
    printf("Execution for N = %d\n", N);
    printf("Serial time: %f\n", t_serial);
    printf("Parallel time: %f\n", t_parallel);

    // Free memory on host
    free(x);
    free(y);

    return 0;

}