
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 4 


__global__ void matrixMult(float *a, float *b, float *c) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;

    for (int k = 0; k < N; k++) {
        sum += a[row * N + k] * b[k * N + col];
    }

    c[row * N + col] = sum;
}

int main() {
    float *h_a, *h_b, *h_c; 
    float *d_a, *d_b, *d_c; 

    h_a = (float*) malloc(N * N * sizeof(float));
    h_b = (float*) malloc(N * N * sizeof(float));
    h_c = (float*) malloc(N * N * sizeof(float));

    for (int i = 0; i < N * N; i++) {
        h_a[i] = i;
        h_b[i] = i + 1;
    }

    printf("Matriz a:\n");
    for (int i = 0; i < N * N; i++) {
        printf("%.2f ", h_a[i]);
        if ((i + 1) % N == 0) {
            printf("\n");
        }
    }
    printf("\n");

    printf("Matriz b:\n");
    for (int i = 0; i < N * N; i++) {
        printf("%.2f ", h_b[i]);
        if ((i + 1) % N == 0) {
            printf("\n");
        }
    }
    printf("\n");

    hipMalloc(&d_a, N * N * sizeof(float));
    hipMalloc(&d_b, N * N * sizeof(float));
    hipMalloc(&d_c, N * N * sizeof(float));

    hipMemcpy(d_a, h_a, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(N, N);
    dim3 numBlocks(1, 1);

    matrixMult<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c);

    hipMemcpy(h_c, d_c, N * N * sizeof(float), hipMemcpyDeviceToHost);

    printf("Matriz c:\n");
    for (int i = 0; i < N * N; i++) {
        printf("%.2f ", h_c[i]);
        if ((i + 1) % N == 0) {
            printf("\n");
        }
    }

    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
